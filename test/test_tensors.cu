#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiptensor.h>
#include <gtest/gtest.h>
#include <stdio.h>
#include <stdlib.h>
#include <tensor.hpp>
#include <unordered_map>
#include <vector>

TEST(SimpleCopyTest, Copy) {
  float *hostPtr;
  float *devicePtr;

  size_t size = 1024 * sizeof(float);

  hostPtr = (float *)malloc(size);
  ASSERT_NE(hostPtr, nullptr) << "Failed to allocate host memory";

  hipError_t err = hipMalloc((void **)&devicePtr, size);
  ASSERT_EQ(err, hipSuccess)
      << "Failed to allocate device memory: " << hipGetErrorString(err);

  hipError_t copyErr =
      hipMemcpy(devicePtr, hostPtr, size, hipMemcpyHostToDevice);
  ASSERT_EQ(copyErr, hipSuccess)
      << "Failed to copy memory from host to device: "
      << hipGetErrorString(copyErr);
}

int getGpuId() {
  int deviceCount;
  hipError_t res;

  if ((res = hipGetDeviceCount(&deviceCount)) != hipSuccess) {
    std::cerr << "Error getting device count: " << hipGetErrorString(res)
              << std::endl;
    return -1;
  }

  if (deviceCount == 0) {
    std::cerr << "No CUDA devices found." << std::endl;
    return -1;
  }

  int gpuId = 0; // Default to the first GPU
  hipSetDevice(gpuId);
  return gpuId;
}

// Tests that we can allocate a tensor and move it between the CPU and the GPU.
TEST(TestTensorAlloc, Size) {
  std::vector<size_t> shape = {3, 4};

  // Get some CUDA devices
  int gpuId = getGpuId();
  ASSERT_GE(gpuId, 0) << "Failed to get a valid GPU ID";

  float *data = (float *)malloc(sizeof(float) * 12);
  ASSERT_NE(data, nullptr) << "Failed to allocate host memory for data";

  Device cpu = {0, DeviceType::CPU};
  Tensor tens(shape, cpu);

  Device gpu = {gpuId, DeviceType::CUDA};
  tens.moveToDevice(gpu);
  printf("Tensor moved to GPU %d\n", gpuId);

  tens.moveToDevice(cpu);
  printf("Tensor moved back to CPU\n");

  free(data);
}
