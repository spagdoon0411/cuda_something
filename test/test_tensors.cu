#include <gtest/gtest.h>
#include <tensor.hpp>

TEST(SimpleCopyTest, Copy) {
  float *hostPtr;
  float *devicePtr;

  size_t size = 1024 * sizeof(float);

  hostPtr = (float *)malloc(size);
  ASSERT_NE(hostPtr, nullptr) << "Failed to allocate host memory";

  hipError_t err = hipMalloc((void **)&devicePtr, size);
  ASSERT_EQ(err, hipSuccess)
      << "Failed to allocate device memory: " << hipGetErrorString(err);

  hipError_t copyErr =
      hipMemcpy(devicePtr, hostPtr, size, hipMemcpyHostToDevice);
  ASSERT_EQ(copyErr, hipSuccess)
      << "Failed to copy memory from host to device: "
      << hipGetErrorString(copyErr);
}

int getGpuId() {
  int deviceCount;
  hipError_t res;

  if ((res = hipGetDeviceCount(&deviceCount)) != hipSuccess) {
    std::cerr << "Error getting device count: " << hipGetErrorString(res)
              << std::endl;
    return -1;
  }

  if (deviceCount == 0) {
    std::cerr << "No CUDA devices found." << std::endl;
    return -1;
  }

  int gpuId = 0; // Default to the first GPU
  hipSetDevice(gpuId);
  return gpuId;
}

TEST(TestTensorAlloc, Size) {
  std::vector<size_t> shape = {3, 4};

  // Get some CUDA devices
  int gpuId = getGpuId();
  ASSERT_GE(gpuId, 0) << "Failed to get a valid GPU ID";

  float *data = (float *)malloc(sizeof(float) * 12);
  ASSERT_NE(data, nullptr) << "Failed to allocate host memory for data";

  Device cpu = {0, DeviceType::CPU};
  Tensor tens(shape, cpu);

  Device gpu = {gpuId, DeviceType::CUDA};
  tens.moveToDevice(gpu);
  printf("Tensor moved to GPU %d\n", gpuId);

  tens.moveToDevice(cpu);
  printf("Tensor moved back to CPU\n");

  free(data);
}
