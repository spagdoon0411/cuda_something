
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int processArguments(int argc, char *argv[]) {
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " <number_of_elements>" << std::endl;
    return -1;
  }
  return atoi(argv[1]);
}

int main(int argc, char *argv[]) {
  int N = 1 << 20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int numThreads = processArguments(argc, argv);

  // Run kernel on 1M elements on the GPU
  add<<<1, numThreads>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  return 0;
}
