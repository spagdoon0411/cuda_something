#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t prop;
    hipError_t res = hipGetDeviceProperties(&prop, i);
    if (res != hipSuccess) {
      printf("Error getting properties for device %d\n", i);
      continue;
    }
    printf("Device %d: %s\n", i, prop.name);
  }
  return 0;
}
