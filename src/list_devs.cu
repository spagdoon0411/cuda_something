#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int i = 0; i < deviceCount; ++i) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device %d: %s\n", i, prop.name);
  }
  return 0;
}
